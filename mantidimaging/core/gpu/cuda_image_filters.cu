
#include <hip/hip_runtime.h>
extern "C" {
__device__ void print_float_array(const float *array, const int N) {
  for (int i = 0; i < N; i++)
    printf("%.3f ", array[i]);
  printf("\n");
}
__device__ void print_neighbour_elements(const float *padded_array,
                                         const int index_offset,
                                         const int padded_img_width,
                                         const int id_x, const int id_y,
                                         const int filter_size) {
  for (int i = id_x; i < id_x + filter_size; i++)
    for (int j = id_y; j < id_y + filter_size; j++)
      printf("%.3f ", padded_array[index_offset + (i * padded_img_width) + j]);
  printf("\n");
}
__device__ float find_median_in_neighbour_array(float *neighbour_array,
                                                const int N) {
  int i, j;
  float key;

  for (i = 1; i < N; i++) {
    key = neighbour_array[i];
    j = i - 1;

    while (j >= 0 && neighbour_array[j] > key) {
      neighbour_array[j + 1] = neighbour_array[j];
      j = j - 1;
    }
    neighbour_array[j + 1] = key;
  }

  return neighbour_array[N / 2];
}
__device__ float find_neighbour_median(const float *padded_array,
                                       const int padded_img_width,
                                       const int id_x, const int id_y,
                                       const int filter_size) {
  float *neighbour_array = new float[filter_size * filter_size];
  int n_counter = 0;

  for (int i = id_x; i < id_x + filter_size; i++) {
    for (int j = id_y; j < id_y + filter_size; j++) {
      neighbour_array[n_counter] = padded_array[(i * padded_img_width) + j];
      n_counter += 1;
    }
  }

  float median = find_median_in_neighbour_array(neighbour_array,
                                                filter_size * filter_size);
  free(neighbour_array);
  return median;
}
__global__ void two_dimensional_median_filter(float *data_array,
                                              const float *padded_array,
                                              const int X, const int Y,
                                              const int filter_size) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  data_array[index] = find_neighbour_median(padded_array, padded_img_width,
                                            id_x, id_y, filter_size);
}
__global__ void two_dimensional_remove_light_outliers(float *data_array,
                                                      const float *padded_array,
                                                      const int X, const int Y,
                                                      const int filter_size,
                                                      const float diff) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  float median = find_neighbour_median(padded_array, padded_img_width, id_x,
                                       id_y, filter_size);

  if (data_array[index] - median >= diff)
    data_array[index] = median;
}
__global__ void two_dimensional_remove_dark_outliers(float *data_array,
                                                     const float *padded_array,
                                                     const int X, const int Y,
                                                     const int filter_size,
                                                     const float diff) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  float median = find_neighbour_median(padded_array, padded_img_width, id_x,
                                       id_y, filter_size);

  if (median - data_array[index] >= diff)
    data_array[index] = median;
}
}
