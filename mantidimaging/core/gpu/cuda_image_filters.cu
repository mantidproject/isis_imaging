#include "hip/hip_runtime.h"
extern "C" {
__device__ void print_float_array(const float *array, const int N) {
  for (int i = 0; i < N; i++)
    printf("%.3f ", array[i]);
  printf("\n");
}
__device__ float print_neighbour_elements(const float *padded_array,
                                          const int index_offset,
                                          const int padded_img_width,
                                          const int id_x, const int id_y,
                                          const int filter_size) {
  for (int i = id_x; i < id_x + filter_size; i++)
    for (int j = id_y; j < id_y + filter_size; j++)
      printf("%.3f ", padded_array[index_offset + (i * padded_img_width) + j]);
  printf("\n");
}
__device__ float find_median_in_one_dim_array(float *neighb_array,
                                              const int N) {
  int i, j;
  float key;

  for (i = 1; i < N; i++) {
    key = neighb_array[i];
    j = i - 1;

    while (j >= 0 && neighb_array[j] > key) {
      neighb_array[j + 1] = neighb_array[j];
      j = j - 1;
    }
    neighb_array[j + 1] = key;
  }

  return neighb_array[N / 2];
}
__device__ float find_neighbour_median(const float *padded_array,
                                       const int index_offset,
                                       const int padded_img_width,
                                       const int id_x, const int id_y,
                                       const int filter_size) {
  float neighb_array[25];
  int n_counter = 0;

  for (int i = id_x; i < id_x + filter_size; i++) {
    for (int j = id_y; j < id_y + filter_size; j++) {
      neighb_array[n_counter] =
          padded_array[index_offset + (i * padded_img_width) + j];
      n_counter += 1;
    }
  }

  return find_median_in_one_dim_array(neighb_array, filter_size * filter_size);
}
__global__ void image_stack_median_filter(float *data_array,
                                          const float *padded_array,
                                          const int N_IMAGES, const int X,
                                          const int Y, const int filter_size) {
  unsigned int id_img = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_x = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int id_y = blockIdx.z * blockDim.z + threadIdx.z;

  if ((id_img >= N_IMAGES) || (id_x >= X) || (id_y >= Y))
    return;

  unsigned int img_size = X * Y;
  unsigned int padded_img_width = Y + filter_size - 1;
  unsigned int padded_img_size = padded_img_width * (X + filter_size - 1);

  data_array[(id_img * img_size) + (id_x * X) + id_y] =
      find_neighbour_median(padded_array, id_img * padded_img_size,
                            padded_img_width, id_x, id_y, filter_size);
}
__global__ void two_dim_median_filter(float *data_array,
                                      const float *padded_array, const int X,
                                      const int Y, const int filter_size) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int padded_img_width = Y + filter_size - 1;
  unsigned int index = (id_x * Y) + id_y;

  data_array[index] = find_neighbour_median(padded_array, 0, padded_img_width,
                                            id_x, id_y, filter_size);
}
__global__ void three_dim_async_median_filter(float *data_array,
                                              const float *padded_array,
                                              const int X, const int Y,
                                              const int filter_size) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int padded_img_height = X + filter_size - 1;
  unsigned int padded_img_width = Y + filter_size - 1;
  unsigned int index = ((filter_size / 2) * X * Y) + (id_x * Y) + id_y;
  unsigned int n_counter = 0;
  float neighb_array[27];

  for (int i = id_x; i < id_x + filter_size; i++) {
    for (int j = id_y; j < id_y + filter_size; j++) {
      neighb_array[n_counter] = data_array[];
      n_counter++;
    }
  }

  data_array[index] = find_median_in_one_dim_array(
      neighb_array, filter_size * filter_size * filter_size);
}
__global__ void two_dim_remove_light_outliers(float *data_array,
                                              const float *padded_array,
                                              const int X, const int Y,
                                              const int filter_size,
                                              const float diff) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  float median = find_neighbour_median(padded_array, 0, padded_img_width, id_x,
                                       id_y, filter_size);

  if (data_array[index] - median >= diff)
    data_array[index] = median;
}
__global__ void two_dim_remove_dark_outliers(float *data_array,
                                             const float *padded_array,
                                             const int X, const int Y,
                                             const int filter_size,
                                             const float diff) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  float median = find_neighbour_median(padded_array, 0, padded_img_width, id_x,
                                       id_y, filter_size);

  if (median - data_array[index] >= diff)
    data_array[index] = median;
}
}
