
#include <hip/hip_runtime.h>
extern "C" {
/**
  Prints a float array. Can be helpful for debugging.

  @param array     The float array.
  @param N         The size of the array.
 */
__device__ void print_float_array(const float *array, const int N) {
  for (int i = 0; i < N; i++)
    printf("%.3f ", array[i]);
  printf("\n");
}
/**
  Prints the neighbour elements of a pixel in a 2D array. Can be helpful for
  debugging.

  @param padded_array        The padded data array.
  @param padded_img_width    The width of the padded image.
  @param id_x                The x index of the current pixel.
  @param id_y                The y index of the current pixel.
  @param filter_size         The size of the filter.
 */
__device__ void print_neighbour_elements_in_two_dimensional_array(
    const float *padded_array, const int padded_img_width, const int id_x,
    const int id_y, const int filter_size) {
  for (int i = id_x; i < id_x + filter_size; i++)
    for (int j = id_y; j < id_y + filter_size; j++)
      printf("%.3f ", padded_array[(i * padded_img_width) + j]);
  printf("\n");
}
/**
  Insertion sorts a 1D array and returns its median.
  Helper function for the 2D median and 2D remove outlier filters.

  @param array     The float array.
  @param N         The size of the array.
  @return          The median of the array.
 */
__device__ float find_median_in_neighbour_array(float *neighbour_array,
                                                const int N) {
  int i, j;
  float key;

  for (i = 1; i < N; i++) {
    key = neighbour_array[i];
    j = i - 1;

    while (j >= 0 && neighbour_array[j] > key) {
      neighbour_array[j + 1] = neighbour_array[j];
      j = j - 1;
    }
    neighbour_array[j + 1] = key;
  }

  return neighbour_array[N / 2];
}
/**
  Returns the median of a pixel's neighbours in a 2D array.
  Helper function for the 2D median and 2D remove outlier filters.

  @param padded_array        The padded data array.
  @param padded_img_width    The width of the padded image.
  @param id_x                The x index of the current pixel.
  @param id_y                The y index of the current pixel.
  @param filter_size         The size of the filter.
  @return                    The median of the pixel's neighbourhood.
 */
__device__ float find_neighbour_median(const float *padded_array,
                                       const int padded_img_width,
                                       const int id_x, const int id_y,
                                       const int filter_size) {
  float *neighbour_array = new float[filter_size * filter_size];
  int n_counter = 0;

  for (int i = id_x; i < id_x + filter_size; i++) {
    for (int j = id_y; j < id_y + filter_size; j++) {
      neighbour_array[n_counter] = padded_array[(i * padded_img_width) + j];
      n_counter += 1;
    }
  }

  float median = find_median_in_neighbour_array(neighbour_array,
                                                filter_size * filter_size);
  free(neighbour_array);
  return median;
}
/**
  Applies a median filter to all the pixels in a 2D array.
  This function should be used asynchronously with a stack of 2D images.

  @param data_array       The original data array.
  @param padded_array     The padded data array.
  @param X                The height of the image.
  @param Y                The width of the image.
  @param filter_size      The size of the filter.
 */
__global__ void two_dimensional_median_filter(float *data_array,
                                              const float *padded_array,
                                              const int X, const int Y,
                                              const int filter_size) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  data_array[index] = find_neighbour_median(padded_array, padded_img_width,
                                            id_x, id_y, filter_size);
}
/**
  Applies a remove light outliers filter to all the pixels in a 2D array.
  This function should be used asynchronously with a stack of 2D images.

  @param data_array       The original data array.
  @param padded_array     The padded data array.
  @param X                The height of the image.
  @param Y                The width of the image.
  @param filter_size      The size of the filter.
  @param diff             The difference required to replace the original pixel
                          value with the median.
 */
__global__ void two_dimensional_remove_light_outliers(float *data_array,
                                                      const float *padded_array,
                                                      const int X, const int Y,
                                                      const int filter_size,
                                                      const float diff) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  float median = find_neighbour_median(padded_array, padded_img_width, id_x,
                                       id_y, filter_size);

  if (data_array[index] - median >= diff)
    data_array[index] = median;
}
/**
  Applies a remove light outliers filter to all the pixels in a 2D array.
  This function should be used asynchronously with a stack of 2D images.

  @param data_array       The original data array.
  @param padded_array     The padded data array.
  @param X                The height of the image.
  @param Y                The width of the image.
  @param filter_size      The size of the filter.
  @param diff             The difference required to replace the original pixel
                          value with the median.
 */
__global__ void two_dimensional_remove_dark_outliers(float *data_array,
                                                     const float *padded_array,
                                                     const int X, const int Y,
                                                     const int filter_size,
                                                     const float diff) {
  unsigned int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int id_y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((id_x >= X) || (id_y >= Y))
    return;

  unsigned int index = (id_x * Y) + id_y;
  unsigned int padded_img_width = Y + filter_size - 1;

  float median = find_neighbour_median(padded_array, padded_img_width, id_x,
                                       id_y, filter_size);

  if (median - data_array[index] >= diff)
    data_array[index] = median;
}
}
